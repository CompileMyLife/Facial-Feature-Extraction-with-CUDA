#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel() {
    // do nothing
}

int main(){
    kernel<<<1,1>>>();
    hipDeviceSynchronize();
    printf("Hello from CUDA!\n");
    return 0;
}
